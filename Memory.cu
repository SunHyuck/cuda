#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>




int main(void) {
    const int SIZE = 8;
    const float a[SIZE] = {1., 2., 3., 4., 5., 6., 7., 8.};
    float b[SIZE] = {0., 0., 0., 0., 0., 0., 0., 0.};

    printf("a = {%f, %f, %f, %f, %f, %f, %f, %f}\n", a[0], a[1], a[2], a[3], a[4], a[5], a[6], a[7]);
    printf("Before b = {%f, %f, %f, %f, %f, %f, %f, %f}\n", b[0], b[1], b[2], b[3], b[4], b[5], b[6], b[7]);

    float* dev_a = nullptr;
    float* dev_b = nullptr;

    hipMalloc((void**)&dev_a, SIZE*sizeof(float));
    hipMalloc((void**)&dev_b, SIZE*sizeof(float));

    hipMemcpy(dev_a, a, SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, dev_a, SIZE*sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(b, dev_b, SIZE*sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree(dev_a);
    hipFree(dev_b);

    printf("After b = {%f, %f, %f, %f, %f, %f, %f, %f}\n", b[0], b[1], b[2], b[3], b[4], b[5], b[6], b[7]);
    return 0;
}