#include "hip/hip_runtime.h"
#include "./common.cpp"
#include <stdio.h>
#include <stdlib.h>

const unsigned vecSize = 256*1024*1024;
float lerp_t = 0.234F;

__global__ void kernel_lerp(float* z, const float t, const float* x, const float* y, unsigned n) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        z[i] = (1.0F-t)*x[i] + t*y[i];
    }
}

__global__ void kernel_lerp_fma(float* z, const float t, const float* x, const float* y, unsigned n) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        z[i] = fmaf(t, y[i], fmaf(-t, x[i], x[i]));
    }
}

int main() {
    float* vecX = new float[vecSize];
    float* vecY = new float[vecSize];
    float* vecZ = new float[vecSize];

    float* dev_vecX = nullptr;
    float* dev_vecY = nullptr;
    float* dev_vecZ = nullptr;

    hipMalloc((void**)&dev_vecX, vecSize*sizeof(float));
    hipMalloc((void**)&dev_vecY, vecSize*sizeof(float));
    hipMalloc((void**)&dev_vecZ, vecSize*sizeof(float));
    CUDA_CHECK_ERROR();

    srand(0);
    setNormalizedRandomData(vecX, vecSize);
    setNormalizedRandomData(vecY, vecSize);

    ELAPSED_TIME_BEGIN(3);

    hipMemcpy(dev_vecX, vecX, vecSize*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_vecY, vecY, vecSize*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    dim3 dimBlock(1024, 1, 1);
    dim3 dimGrid((vecSize+dimBlock.x-1)/dimBlock.x, 1, 1);
    ELAPSED_TIME_BEGIN(1);
    kernel_lerp<<<dimGrid,dimBlock>>>(dev_vecZ, lerp_t, dev_vecX, dev_vecY, vecSize);
    hipDeviceSynchronize();
    ELAPSED_TIME_END(1);
    CUDA_CHECK_ERROR();

    ELAPSED_TIME_BEGIN(2);
    kernel_lerp_fma<<<dimGrid,dimBlock>>>(dev_vecZ, lerp_t, dev_vecX, dev_vecY, vecSize);
    hipDeviceSynchronize();
    ELAPSED_TIME_END(2);
    CUDA_CHECK_ERROR();

    hipMemcpy(vecZ, dev_vecZ, vecSize*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK_ERROR();
    ELAPSED_TIME_END(3);

    hipFree(dev_vecX);
    hipFree(dev_vecY);
    hipFree(dev_vecZ);

    float sumX = getSum(vecX, vecSize);
    float sumY = getSum(vecY, vecSize);
    float sumZ = getSum(vecZ, vecSize);
    float diff = fabsf(sumZ - ((1-lerp_t)*sumX+lerp_t*sumY));

    printf("vecSize = %d\n", vecSize);
    printf("t    = %f\n", lerp_t);
    printf("sumX = %f\n", sumX);
    printf("sumY = %f\n", sumY);
    printf("sumZ = %f\n", sumZ);
    printf("diff(sumZ, (1-t)*sumX+t*sumY) = %f\n", diff);
    printf("diff(sumZ, (1-t)*sumX+t*sumY)/vecSize = %f\n", diff/vecSize);

    printf("vecX = [%8f %8f %8f %8f ... %8f %8f %8f %8f]\n", vecX[0], vecX[1], vecX[2], vecX[3], vecX[vecSize-4], vecX[vecSize-3], vecX[vecSize-2], vecX[vecSize-1]);
    printf("vecY = [%8f %8f %8f %8f ... %8f %8f %8f %8f]\n", vecY[0], vecY[1], vecY[2], vecY[3], vecY[vecSize-4], vecY[vecSize-3], vecY[vecSize-2], vecY[vecSize-1]);
    printf("vecZ = [%8f %8f %8f %8f ... %8f %8f %8f %8f]\n", vecZ[0], vecZ[1], vecZ[2], vecZ[3], vecZ[vecSize-4], vecZ[vecSize-3], vecZ[vecSize-2], vecZ[vecSize-1]);


    delete[] vecX;
    delete[] vecY;
    delete[] vecZ;
}